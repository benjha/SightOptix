#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"
using namespace optix;


struct BasicLight
{
#if defined(__cplusplus)
  typedef optix::float3 float3;
#endif
  float3 pos;
  float3 color;
  int    casts_shadow;
  int    padding;      // make this structure 32 bytes -- powers of two are your friend!
};

struct PerRayData_radiance
{
  float3 result;
  float3 albedo;
  float3 normal;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};


rtDeclareVariable(float3, 		bg_color, , );
rtDeclareVariable(float3,       eye, , );
rtDeclareVariable(float3,       U, , );
rtDeclareVariable(float3,       V, , );
rtDeclareVariable(float3,       W, , );
rtDeclareVariable(float4,       bad_color, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,		top_object, , );
rtDeclareVariable(unsigned int,	radiance_ray_type, , );
rtDeclareVariable(unsigned int,	shadow_ray_type, , );
rtDeclareVariable(float3, 		shading_color, attribute shading_color, );


rtDeclareVariable(uint2, 		launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, 		launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, 		time_view_scale, , ) = 1e-6f;
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// for antialiasing
rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(unsigned int,	 frame, , );
rtDeclareVariable(float,         jitter_factor, ,) = 0.0f;

rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>				 denoised_buffer;
rtBuffer<uchar4, 2>				 bufferToEncode;
rtBuffer<float4, 2>              accum_buffer;
rtBuffer<float4, 2>              albedo_buffer;
rtBuffer<float4, 2>              normal_buffer;



//#define TIME_VIEW

RT_PROGRAM void pinhole_camera()
{
#ifdef TIME_VIEW
	clock_t t0 = clock();
#endif
	size_t2 screen = output_buffer.size();
	// Subpixel jitter: send the ray through a different position inside the pixel each time,
	// to provide antialiasing.
	unsigned int seed = rot_seed( rnd_seeds[ launch_index ], frame );
	float2 subpixel_jitter = make_float2(rnd( seed ) - 0.5f, rnd( seed ) - 0.5f) * jitter_factor;
	float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

    float3 albedo = make_float3(0.0f);
    float3 normal = make_float3(0.0f);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX );

	PerRayData_radiance prd;
    prd.result = make_float3(0.0f);
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

#ifdef TIME_VIEW
	clock_t t1 = clock();

	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
	output_buffer[launch_index] = make_color( make_float3(  pixel_time ) );
#else
	float4 acc_val = accum_buffer[launch_index];
	if( frame > 1 ){
		acc_val = lerp( acc_val, make_float4( prd.result, 0.f), 1.0f / static_cast<float>( frame+1 ) );

	}
	else
		acc_val = make_float4(prd.result, 0.f);

	output_buffer[launch_index] = acc_val;
	accum_buffer[launch_index] = acc_val;
	albedo_buffer[launch_index] = make_float4(prd.albedo, 1.0f);
	normal_buffer[launch_index] = make_float4(prd.normal,1.0f);

#endif
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = bad_color;
}

RT_PROGRAM void exception2()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  denoised_buffer[launch_index] = bad_color;
}


RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;

	if (prd_radiance.depth==0)
	{
		prd_radiance.albedo = bg_color;
		prd_radiance.normal = make_float3(0.0f,0.0f,0.0f);
	}
}
// no denoising
RT_PROGRAM void float4TOcolor ()
{
	bufferToEncode[launch_index] = make_color (make_float3(output_buffer[launch_index]));
}

// when denoising is used
RT_PROGRAM void float4TOcolorDenoisedBuffer ()
{
	bufferToEncode[launch_index] = make_color (make_float3(denoised_buffer[launch_index]));
}


// AO
//=======================================================================================
//


struct PerRayData_occlusion
{
  float occlusion;
};

rtDeclareVariable(PerRayData_occlusion, prd_occlusion, rtPayload, );

rtDeclareVariable(float,       occlusion_distance, , );
rtDeclareVariable(int,         sqrt_occlusion_samples, , );

rtDeclareVariable(optix::Ray,	ray,          		rtCurrentRay, );
rtDeclareVariable(float,      	t_hit,        		rtIntersectionDistance, );
rtDeclareVariable(unsigned int, subframe_idx, 		rtSubframeIndex, );
rtDeclareVariable(float3,		geometric_normal, 	attribute geometric_normal, );
rtDeclareVariable(float3, 		shading_normal, 	attribute shading_normal, );

RT_PROGRAM void closest_hit_radiance_AO()
{
  float3 phit    = ray.origin + t_hit * ray.direction;

  float3 world_shading_normal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

  optix::Onb onb(ffnormal);

  unsigned int seed = rot_seed(rnd_seeds[launch_index], frame + subframe_idx);

  float       result           = 0.0f;
  const float inv_sqrt_samples = 1.0f / float(sqrt_occlusion_samples);
  for( int i=0; i<sqrt_occlusion_samples; ++i ) {
    for( int j=0; j<sqrt_occlusion_samples; ++j ) {

      PerRayData_occlusion prd_occ;
      prd_occ.occlusion = 0.0f;

      // Stratify samples via simple jitterring
      float u1 = (float(i) + rnd( seed ) )*inv_sqrt_samples;
      float u2 = (float(j) + rnd( seed ) )*inv_sqrt_samples;

      float3 dir;
      optix::cosine_sample_hemisphere( u1, u2, dir );
      onb.inverse_transform( dir );

      optix::Ray occlusion_ray = optix::make_Ray( phit, dir, 1, scene_epsilon,
                                                  occlusion_distance );
      rtTrace( top_object, occlusion_ray, prd_occ );

      result += 1.0f-prd_occ.occlusion;
    }
  }

  result /= (float)(sqrt_occlusion_samples*sqrt_occlusion_samples);

  prd_radiance.result = make_float3(result) * shading_color;
}

RT_PROGRAM void any_hit_occlusion_AO()
{
  prd_occlusion.occlusion = 1.0f;

  rtTerminateRay();
}


// phong Illumination
//=======================================================================================
//

rtDeclareVariable(int,              max_depth, , );
rtDeclareVariable(float3,           ambient_light_color, , );


rtDeclareVariable(rtObject,			top_shadower, , );

rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float3,       Ka, , );
rtDeclareVariable(float3,       Kd, , );
rtDeclareVariable(float3,       Ks, , );
rtDeclareVariable(float3,       Kr, , );
rtDeclareVariable(float,        phong_exp, , );



rtBuffer<BasicLight>                 lights;

static __device__ void phongShadowed()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = optix::make_float3(0.0f);
  rtTerminateRay();
}

static
__device__ void phongShade( float3 p_Kd,
                            float3 p_Ka,
                            float3 p_Ks,
                            float3 p_Kr,
                            float  p_phong_exp,
                            float3 p_normal )
{
  float3 hit_point = ray.origin + t_hit * ray.direction;

  // ambient contribution
  float3 result = p_Ka * ambient_light_color;

  // compute direct lighting
  unsigned int num_lights = lights.size();
  for(int i = 0; i < num_lights; ++i) {
    BasicLight light = lights[i];
    float Ldist = optix::length(light.pos - hit_point);
    float3 L = optix::normalize(light.pos - hit_point);
    float nDl = optix::dot( p_normal, L);



    // cast shadow ray
    float3 light_attenuation = make_float3(static_cast<float>( nDl > 0.0f ));
    if ( nDl > 0.0f && light.casts_shadow ) {
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      optix::Ray shadow_ray = optix::make_Ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      light_attenuation = shadow_prd.attenuation;
    }

    // If not completely shadowed, light the hit point
    if( fmaxf(light_attenuation) > 0.0f ) {
      float3 Lc = light.color * light_attenuation;

      result += p_Kd * nDl * Lc;

      float3 H = optix::normalize(L - ray.direction);
      float nDh = optix::dot( p_normal, H );
      if(nDh > 0) {
        float power = pow(nDh, p_phong_exp);
        result += p_Ks * power * Lc;
      }
    }
  }

  if( fmaxf( p_Kr ) > 0 ) {

    // ray tree attenuation
    PerRayData_radiance new_prd;
    new_prd.importance = prd.importance * optix::luminance( p_Kr );
    new_prd.depth = prd.depth + 1;

    // reflection ray
    if( new_prd.importance >= 0.01f && new_prd.depth <= max_depth) {
      float3 R = optix::reflect( ray.direction, p_normal );
      optix::Ray refl_ray = optix::make_Ray( hit_point, R, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX );
      rtTrace(top_object, refl_ray, new_prd);
      result += p_Kr * new_prd.result;
    }
  }

  // pass the color back up the tree
  prd.result = result * shading_color;
}



RT_PROGRAM void any_hit_shadow()
{
  phongShadowed();
}


RT_PROGRAM void closest_hit_radiance()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
  phongShade( Kd, Ka, Ks, Kr, phong_exp, ffnormal );
  //prd.result = shading_color;
}


RT_PROGRAM void closest_hit_radiance_AO_phong()
{
  float3 phit    = ray.origin + t_hit * ray.direction;

  float3 world_shading_normal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

  optix::Onb onb(ffnormal);

  // The albedo buffer should contain an approximation of the overall surface albedo (i.e. a single
  // color value approximating the ratio of irradiance reflected to the irradiance received over the
  // hemisphere). This can be approximated for very simple materials by using the diffuse color of
  // the first hit.
  if (prd_radiance.depth == 0)
  {
	  prd_radiance.albedo = shading_color;
	  prd_radiance.normal = ffnormal;
  }

  unsigned int seed = rot_seed(rnd_seeds[launch_index], frame + subframe_idx);

  float       result           = 0.0f;
  const float inv_sqrt_samples = 1.0f / float(sqrt_occlusion_samples);
  for( int i=0; i<sqrt_occlusion_samples; ++i ) {
    for( int j=0; j<sqrt_occlusion_samples; ++j ) {

      PerRayData_occlusion prd_occ;
      prd_occ.occlusion = 0.0f;

      // Stratify samples via simple jitterring
      float u1 = (float(i) + rnd( seed ) )*inv_sqrt_samples;
      float u2 = (float(j) + rnd( seed ) )*inv_sqrt_samples;

      float3 dir;
      optix::cosine_sample_hemisphere( u1, u2, dir );
      onb.inverse_transform( dir );

      optix::Ray occlusion_ray = optix::make_Ray( phit, dir, 1, scene_epsilon,
                                                  occlusion_distance );
      rtTrace( top_object, occlusion_ray, prd_occ );

      result += 1.0f-prd_occ.occlusion;
    }
  }

  result /= (float)(sqrt_occlusion_samples*sqrt_occlusion_samples);

  // Phong
  phongShade( Kd, Ka, Ks, Kr, phong_exp, ffnormal );

  // Phong result in prd.result
  prd_radiance.result = make_float3(result);

  prd_radiance.result.x *= prd.result.x;
  prd_radiance.result.y *= prd.result.y;
  prd_radiance.result.z *= prd.result.z;
}




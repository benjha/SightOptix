#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "../header/sightOptixStructs.h"
//#define	COLOR_LOOK_UP

// How to compile cu files for optix
//  /usr/local/cuda-8.0/bin/nvcc -I../frameserver/header -I../frameserver/communications -I../../../Programs/NVIDIA-OptiX-SDK-5.0.1-linux64/include -O3 -std=c++11 -gencode arch=compute_35,code=sm_35 -gencode -gencode arch=compute_70,code=compute_70 -gencode arch=compute_50,code=sm_50 -ptx -m64 spheres.cu


using namespace optix;

rtBuffer<float4>	particle_buffer;
rtBuffer<sightSphereColor> sphere_buffer;

#ifndef COLOR_LOOK_UP
	rtBuffer<float4>	color_buffer;
#endif

//rtDeclareVariable(float4,  sphere, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, shading_color, attribute shading_color, );


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

#ifdef COLOR_LOOK_UP
static __device__
float3 colorLookUp (float idx)
{
	if (idx==0.0f)
	{
		return make_float3(1.0f, 0.0f, 0.0f);
	}
	else if (idx==1.0f)
	{
		return make_float3(0.0f, 1.0f, 0.0f);
	}
	else if (idx==2.0f)
	{
		return make_float3(1.0f, 1.0f, 0.0f);
	}
	else if (idx==3.0f)
	{
		return make_float3(0.0f, 1.0f, 1.0f);
	}
	return make_float3(0.0f, 0.0f, 1.0f);
}
#endif

template<bool use_robust_method>
static __device__
void intersect_sphere(int primIdx)
{
	const float4 center4=sphere_buffer[primIdx].center;
	const float3 color=make_float3(sphere_buffer[primIdx].color);
	const float3 center=make_float3 (center4);
	const float radius = center4.w;

#ifdef COLOR_LOOK_UP
	float 			colorIdx = lookUp.w;
#endif

	float3 O = ray.origin - center;
	float3 D = ray.direction;

	float b = dot(O, D);
	float c = dot(O, O)-radius*radius;
	float disc = b*b-c;

	if(disc > 0.0f)
	{
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);
		bool do_refine = false;
		float root11 = 0.0f;

		if(use_robust_method && fabsf(root1) > 10.f * radius)
		{
			do_refine = true;
		}
		if(do_refine)
		{
			// refine root1
			float3 O1 = O + root1 * ray.direction;
			b = dot(O1, D);
			c = dot(O1, O1) - radius*radius;
			disc = b*b - c;

			if(disc > 0.0f)
			{
				sdisc = sqrtf(disc);
				root11 = (-b - sdisc);
			}
		}
		bool check_second = true;
		if( rtPotentialIntersection( root1 + root11 ) )
		{
			shading_normal 	= geometric_normal = (O + (root1 + root11)*D)/radius;
#ifdef COLOR_LOOK_UP
			shading_color	= colorLookUp((int)colorIdx%4);
#else
			shading_color=color;
#endif
			//shading_color	= make_float3(1.0f, 0.0f, 0.0f);


			if(rtReportIntersection(0))
				check_second = false;
		}
		/*
		if(check_second)
		{
			float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
			if( rtPotentialIntersection( root2 ) )
			{
				shading_normal = geometric_normal = (O + root2*D)/radius;
#ifdef COLOR_LOOK_UP
			shading_color	= colorLookUp((int)colorIdx%4);
#else
			shading_color	= make_float3 (color_buffer[primIdx]);
#endif

				rtReportIntersection(0);
			}
		}
		*/
	}
}


RT_PROGRAM void sphere_array_intersect(int primIdx)
{

	const float4	lookUp 	= particle_buffer[primIdx];
	const float3	center 	= make_float3 (lookUp);
	const float		radius 	= lookUp.w;

	float3 V = center - ray.origin;
	float b = dot(V, ray.direction);
	float disc = b*b + radius*radius - dot(V, V);
	if (disc > 0.0f) {
		disc = sqrtf(disc);

//#define FASTONESIDEDSPHERES 1
#if defined(FASTONESIDEDSPHERES)
	// only calculate the nearest intersection, for speed
    float t1 = b - disc;
		if (rtPotentialIntersection(t1))
		{
			shading_normal = geometric_normal = (t1*ray.direction - V) / radius;
	    	shading_color	= make_float3 (color_buffer[primIdx]);
			rtReportIntersection(0);
		}
 #else
	float t2 = b + disc;
    if (rtPotentialIntersection(t2))
    {
    	shading_normal = geometric_normal = (t2*ray.direction - V) / radius;
    	//shading_color	= make_float3 (color_buffer[primIdx]);

    	rtReportIntersection(0);
    }
    float t1 = b - disc;
    if (rtPotentialIntersection(t1))
    {
    	shading_normal = geometric_normal = (t1*ray.direction - V) / radius;
    	//shading_color	= make_float3 (color_buffer[primIdx]);
    	rtReportIntersection(0);
    }
#endif
	}
}

RT_PROGRAM void intersect(int primIdx)
{
	//rtPrintf( "primIdx %d", primIdx);
	intersect_sphere<false>( primIdx );

}


RT_PROGRAM void robust_intersect(int primIdx)
{
	intersect_sphere<true>( primIdx );
}


RT_PROGRAM void bounds (int primIdx, float result[6])
{
	//rtPrintf( "primIdx %d", primIdx);
//	const int 		idx		= index_buffer[primIdx];
	const float4 center4=sphere_buffer[primIdx].center;
	const float3 center=make_float3 (center4);
	const float3 vRadius=make_float3 (center4.w);

	optix::Aabb* aabb = (optix::Aabb*)result;

	if ( vRadius.x > 0.0f && !isinf(vRadius.x))
	{
	    aabb->m_min = center - vRadius;
	    aabb->m_max = center + vRadius;
	}
	else
	{
		aabb->invalidate();
	}

}



// 1D buffer
rtBuffer<rtBufferId<float4> > posBufferIds;
rtBuffer<rtBufferId<float4> > colBufferIds;



RT_PROGRAM void robust_intersect_BoB(int primIdx)
{
	unsigned int i;
	for ( i=0; i<posBufferIds.size();++i )
	{
	    // Grab a reference to the nested buffer so we dont need to perform
	    // the buffer lookup multiple times
	    rtBufferId<float4, 1>& posBuffer = posBufferIds[i];

		intersect_sphere<true>( primIdx );
	}
}


